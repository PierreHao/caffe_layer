#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/swish_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SwishForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] / (1. + exp(-in[index]));
  }
}

template <typename Dtype>
void SwishLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SwishForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void SwishBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff, const Dtype* in_data) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype Xsigmoid_x = out_data[index];
    const Dtype sigmoid_x = 1. / (1. + exp(-in_data[index]));
    out_diff[index] = in_diff[index] * ( (Xsigmoid_x + sigmoid_x * (1. - Xsigmoid_x)) );
  }
}

template <typename Dtype>
void SwishLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SwishBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, bottom_diff, bottom_data);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SwishLayer);


}  // namespace caffe
